#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_function_table_definition.h>

#include <hip/hip_runtime.h>
#include <cuda/helpers.h>

#include <stdio.h>
#include <thrust/complex.h>

#include "TriangleSBTData.h"

struct Angle {
	double x;
	double y;
};
struct Point {
	double x;
	double y;
	double z;
};

struct Payload {
	float3 colour;
	float diff;
	float t;
};

extern "C" {
	__constant__ TriangleParams params;
}

extern "C" __global__ void __raygen__rg() {

	const uint3 launchIdx = optixGetLaunchIndex();
	const uint3 launchDim = optixGetLaunchDimensions();

	const RayGenData* rtData = (RayGenData*)optixGetSbtDataPointer();

	const float left = rtData->left;
	const float top = rtData->top;
	const float sizex = rtData->sizex;
	const float sizey = rtData->sizey;
	const float camDist = rtData->camDist;

	const int diamond = rtData->diamond;
	const int xMax = rtData->xMax;
	const int yMax = rtData->yMax;

	const float k = rtData->k;


	float x = left + launchIdx.x * sizex;
	if (diamond && ((launchIdx.y % 2) == 0)) {
		x += sizex / 2.0f;
	}

	float y = top - launchIdx.y * sizey;

	Payload payload;
	payload.colour = make_float3(0.0f);
	payload.diff = 0.0f;
	payload.t = -1;

	unsigned int p0, p1, p2, p3, p4;

	//Compute the reference of the sphere
	//NOTE: thrust::complex example(realNumber, ImagNumber);

	thrust::complex<float> finalComplex_1(0.0, 0.0);
	thrust::complex<float> finalComplex_2(0.0, 0.0);
	thrust::complex<float> finalComplex_3(0.0, 0.0);

	Ray7 currentRay;
	currentRay.origin.x = x;
	currentRay.origin.y = y;
	currentRay.origin.z = -camDist;
	currentRay.tmax = 1.e27f; // The value of RT_DEFAULT_MAX in older OptiX versions
	float3 dirTotal = make_float3(0.0f);

	int numIntersect = 0;
	for (int i = 0; i < yMax; i++) {
		for (int j = 0; j < xMax; j++) {

			payload.t = -1;

			int index = i * xMax + j;

			double dx = rtData->jitterBuffer[index].x;
			double dy = rtData->jitterBuffer[index].y;
			double dz = 1.0;
			double len = sqrt(dx * dx + dy * dy + dz * dz);

			currentRay.direction.x = dx / len;
			currentRay.direction.y = dy / len;
			currentRay.direction.z = dz / len;

			p0 = float_as_int(payload.colour.x);
			p1 = float_as_int(payload.colour.y);
			p2 = float_as_int(payload.colour.z);
			p3 = float_as_int(payload.diff);
			p4 = float_as_int(payload.t);

			optixTrace(
				params.handle,
				currentRay.origin,
				currentRay.direction,
				0.0f, // Intersection dist (min/max)
				1e16f,
				0.0f, // 0 ray-time = 0 motion blur
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_NONE,
				0, // SBT offset / ray type program id
				0, // SBT stride / total number of ray types
				0, // Miss index / active miss program
				p0, p1, p2, p3, p4
			);

			payload.colour.x = int_as_float(p0);
			payload.colour.y = int_as_float(p1);
			payload.colour.z = int_as_float(p2);
			payload.diff = int_as_float(p3);
			payload.t = int_as_float(p4);

			if (payload.t >= 0.0) {

				payload.t -= camDist;

				thrust::complex<float> complexNum(0.0, (float)(k * payload.t));
				complexNum = thrust::exp<float>(complexNum);

				thrust::complex<float> complexColour = thrust::complex<float>((float)payload.colour.x, 0.0);
				complexNum = (complexColour * complexNum) / thrust::complex<float>((float)payload.t, 0.0);
				finalComplex_1 += complexNum;

				complexColour = thrust::complex<float>((float)payload.colour.y, 0.0);
				complexNum = (complexColour * complexNum) / thrust::complex<float>((float)payload.t, 0.0);
				finalComplex_2 += complexNum;

				complexColour = thrust::complex<float>((float)payload.colour.z, 0.0);
				complexNum = (complexColour * complexNum) / thrust::complex<float>((float)payload.t, 0.0);
				finalComplex_3 += complexNum;

				numIntersect += 1;
			}
		}
	}

	// Can convert this to a single int for slight optimization
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_1.x = finalComplex_1.real();
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_1.y = finalComplex_1.imag();

	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_2.x = finalComplex_2.real();
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_2.y = finalComplex_2.imag();

	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_3.x = finalComplex_3.real();
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_3.y = finalComplex_3.imag();

	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].intersect = numIntersect;
}

extern "C" __global__ void __closesthit__ch() {
	int primitiveIndex = optixGetPrimitiveIndex();
	const TriangleHitGroupData* rtData = (TriangleHitGroupData*)optixGetSbtDataPointer();

	float3 colour = rtData->colourBuffer[primitiveIndex];

	// Calculating per-vertex normals with barycentrics
	int3 tri = params.i_buf[primitiveIndex];

	// This is a wrapper function that can be bypassed: https://raytracing-docs.nvidia.com/optix7/api/html/group__optix__device__api.html#gaa121732a59322e799fa026d0997094d4
	float2 bary_temp = optixGetTriangleBarycentrics();
	float3 barycentrics = { bary_temp.x, bary_temp.y, 1-bary_temp.x-bary_temp.y };

	float3 N = (rtData->normals[tri.x] * barycentrics.x)
		+ (rtData->normals[tri.y] * barycentrics.y)
		+ (rtData->normals[tri.z] * barycentrics.z);

	float mag = sqrt(pow(N.x, 2) + pow(N.y, 2) + pow(N.z, 2));
	N.x /= mag; N.y /= mag; N.z /= mag;

	float3 L = optixGetWorldRayDirection();

	mag = sqrt(pow(L.x, 2) + pow(L.y, 2) + pow(L.z, 2));
	L.x /= mag; L.y /= mag; L.z /= mag;

	float diff = N.x*L.x + N.y*L.y + N.z*L.z;

	float ambient = 0.2;
	float diffuse = 0.8;
	float3 result = (diff * diffuse * colour) + ambient;

	optixSetPayload_0(float_as_int(result.x));
	optixSetPayload_1(float_as_int(result.y));
	optixSetPayload_2(float_as_int(result.z));

	optixSetPayload_4(float_as_int(optixGetRayTmax()));		
}
