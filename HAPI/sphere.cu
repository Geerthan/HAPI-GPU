#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_function_table_definition.h>

#include <hip/hip_runtime.h>
#include <cuda/helpers.h>

#include <stdio.h>
#include <thrust/complex.h>

#include "SphereSBTData.h"

struct Angle {
	double x;
	double y;
};
struct Point {
	double x;
	double y;
	double z;
};

struct Payload {
	float3 colour;
	float diff;
	float t;
};

extern "C" {
	__constant__ SphereParams params;
}

extern "C" __global__ void __raygen__rg() {

	const uint3 launchIdx = optixGetLaunchIndex();
	const uint3 launchDim = optixGetLaunchDimensions();

	const RayGenData* rtData = (RayGenData*)optixGetSbtDataPointer();

	const float left	= rtData->left;
	const float top		= rtData->top;
	const float sizex	= rtData->sizex;
	const float sizey	= rtData->sizey;
	const float camDist	= rtData->camDist;

	const int diamond	= rtData->diamond;
	const int xMax		= rtData->xMax;
	const int yMax		= rtData->yMax;

	const float k		= rtData->k;


	float x = left + launchIdx.x * sizex;
	if (diamond && ((launchIdx.y % 2) == 0)) {
		x += sizex / 2.0f;
	}

	float y = top - launchIdx.y * sizey;

	Payload payload;
	payload.colour = make_float3(0.0f);
	payload.diff = 0.0f;
	payload.t = 0.0f;
	unsigned int p0, p1, p2, p3, p4;

	//Compute the reference of the sphere
	//NOTE: thrust::complex example(realNumber, ImagNumber);
	thrust::complex<float> finalComplex(0.0, 0.0);

	Ray7 currentRay;
	currentRay.origin.x = x;
	currentRay.origin.y = y;
	currentRay.origin.z = -camDist;
	currentRay.tmax = 1.e27f; // The value of RT_DEFAULT_MAX
	float3 dirTotal = make_float3(0.0f);

	int numIntersect = 0;
	for (int i = 0; i < xMax; i++) {
		for (int j = 0; j < yMax; j++) {

			int index = i * xMax + j;

			double dx = rtData->jitterBuffer[index].x;
			double dy = rtData->jitterBuffer[index].y;
			double dz = 1.0;
			double len = sqrt(dx * dx + dy * dy + dz * dz);

			currentRay.direction.x = dx / len;
			currentRay.direction.y = dy / len;
			currentRay.direction.z = dz / len;

			p0 = float_as_int(payload.colour.x);
			p1 = float_as_int(payload.colour.y);
			p2 = float_as_int(payload.colour.z);
			p3 = float_as_int(payload.diff);
			p4 = float_as_int(payload.t);

			optixTrace(
				params.handle,
				currentRay.origin,
				currentRay.direction,
				0.0f, // Intersection dist (min/max)
				1e16f,
				0.0f, // 0 ray-time = 0 motion blur
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_NONE,
				0, // SBT offset / ray type program id
				0, // SBT stride / total number of ray types
				0, // Miss index / active miss program
				p0, p1, p2, p3, p4
			);

			payload.colour.x = int_as_float(p0);
			payload.colour.y = int_as_float(p1);
			payload.colour.z = int_as_float(p2);
			payload.diff = int_as_float(p3);
			payload.t = int_as_float(p4);

			if (payload.t >= 0.0) {

				payload.t -= camDist;

				thrust::complex<float> complexNum(0.0, (float)(k * payload.t));
				complexNum = thrust::exp<float>(complexNum);

				thrust::complex<float> complexColour = thrust::complex<float>((float)payload.colour.x, 0.0);
				complexNum = (complexColour * complexNum) / thrust::complex<float>((float)payload.t, 0.0);
				finalComplex += complexNum;

				numIntersect += 1;
			}
		}
	}

	// Can convert this to a single int for slight optimization
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_1.x = finalComplex.real();
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].result_1.y = finalComplex.imag();
	params.image_buffer[launchIdx.y * launchDim.x + launchIdx.x].intersect = numIntersect;
}

extern "C" __global__ void __closesthit__ch() {
	float diff = int_as_float(optixGetAttribute_0());
	float colour = int_as_float(optixGetAttribute_1());

	float ambient = 0.2;
	float diffuse = 0.8;
	float result = diff * diffuse * colour + ambient;

	float3 payloadCol = make_float3(result);
	optixSetPayload_0(float_as_int(payloadCol.x));
	optixSetPayload_1(float_as_int(payloadCol.y));
	optixSetPayload_2(float_as_int(payloadCol.z));
}

extern "C" __global__ void __intersection__sp(int primitiveIndex) {

	primitiveIndex = 0;

	const SphereHitGroupData* rtData = (SphereHitGroupData*)optixGetSbtDataPointer();

	float3 rayOrigin = optixGetWorldRayOrigin();
	float3 rayDirection = optixGetWorldRayDirection();

	//Sphere intersection
	float3 point = rtData->pointBuffer[primitiveIndex];
	double radius = rtData->radiusBuffer[primitiveIndex];

	//Quadratic equation
	double A = 1.0;
	double B;
	double C;
	float3 ec;
	
	//Calculate the discriminant of the quadratic equation
	ec.x = rayOrigin.x - point.x;
	ec.y = rayOrigin.y - point.y;
	ec.z = rayOrigin.z - point.z;
	B = 2 * (rayDirection.x * ec.x + rayDirection.y * ec.y + rayDirection.z * ec.z);

	C = ec.x * ec.x + ec.y * ec.y + ec.z * ec.z - radius * radius;
	double discriminant = B * B - 4.0 * A * C;
	
	optixSetPayload_0(float_as_int(0.0f));
	optixSetPayload_1(float_as_int(0.0f));
	optixSetPayload_2(float_as_int(0.0f));
	optixSetPayload_4(float_as_int(-1.0f));

	float lx = 0.0;
	float ly = 0.0;
	float lz = -0.5;
	float len = sqrt(lx * lx + ly * ly + lz * lz);
	lx /= len;
	ly /= len;
	lz /= len; //lz = -1
	//FIX ME. GET RID OF THE IF STATEMENTS
	float finalT = -1.0;
	if (discriminant >= 0) { //The ray hit the sphere
		discriminant = sqrt(discriminant);
		//two roots of the quadratic equation
		float t1 = (-B - discriminant) / (2 * A);
		float t2 = (-B + discriminant) / (2 * A);
		if (t1 > 0.0f) {
			finalT = t1;
		}
		if (t2 > 0.0f && t2 < t1) {
			finalT = t2;
		}

		float nx = rayOrigin.x + finalT * rayDirection.x - point.x;
		float ny = rayOrigin.y + finalT * rayDirection.y - point.y;
		float nz = rayOrigin.z + finalT * rayDirection.z - point.z;

		len = sqrt(nx * nx + ny * ny + nz * nz);
		nx /= len;
		ny /= len;
		nz /= len;
		float diff = nx * lx + ny * ly + nz * lz;
		if (diff < 0.0) {
			diff = 0.0;
		}
		float colour = rtData->colourBuffer[primitiveIndex];
		
		if (optixReportIntersection(finalT, 0, float_as_int(diff), float_as_int(colour))) {
			optixSetPayload_4(float_as_int(finalT));
		}
	}
}

extern "C" __global__ void __miss__ms() {

}
